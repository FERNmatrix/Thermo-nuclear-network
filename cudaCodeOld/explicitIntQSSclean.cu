
/* Program to calculate explicit QSS or asymptotic integration of a thermonuclear network
 * over a hydrodynamical timestep (temperature and density assumed constant over hydro timestep)
 * using a GPU. The goal is to do the entire network calculation on the GPU by copying the hydro
 * info (temperature, density) and abundances at the end of the hydro timestep from the CPU to
 * the GPU, running the network entirely on the GPU for the duration of the hydro timestep, and then
 * returning the abundances and energy release to the CPU for transmitting to the hydro for the next 
 * timestep, leaving the network intact on the GPU ready for the integration over the next hydro
 * timestep.

Mike Guidry
guidry@utk.edu
July 5, 2012
 
*/

#define BLOCKSIZE 512
#define DIAGNOSE_SIZE 300

// Kernel and device function definitions.  These will run on the device (the GPU).
// There is a max number of arguments and we are close to it. Adding 2-3 more puts us over.

__global__ void integrateNetwork (
	float* P0, 
	float* P1, 
	float* P2, 
	float* P3, 
	float* P4, 
	float* P5, 
	float* P6, 
	float* Prefac,  
	float* Q, 
	float* Rate, 
    float* Flux,
	float* Fplus,
	float* Fminus,
	float* FplusFac,
	float* FminusFac,
	float* FplusSum,
	float* FminusSum, 
 	int* FplusMax,
 	int* FminusMax,
	int* MapFplus,
	int* MapFminus,
	float* Y, 
	float* Diagnose,
	int* Z,               // Not presently used in kernel
	int* N,               // Not presently used in kernel
	int* Params1,
	float* Params2,
	int* NumReactingSpecies,
    int* Reactant1,
	int* Reactant2,
	int* Reactant3
) 
{	
	#define THIRD 0.333333333333333

	// Device function signatures
	__device__ int threadID (void);
	__device__ float computeTimestep (float, float, float);
	__device__ int checkAsy(float, float, float);
	__device__ float asymptoticUpdate(float, float, float, float);
	__device__ float eulerUpdate(float, float, float);
	__device__ float computekeff(float, float);
	
	// Rename integer variables passed in the array Params1
	int numberSpecies = Params1[0];
	int numberReactions = Params1[1];
	int totalFplus = Params1[2];
	int totalFminus = Params1[3];
	
	// Rename float variables passed in the array Params2
	float T9 = Params2[0];
	float tmax = Params2[1];
	float dt_init = Params2[2];

	// Logical control parameters	
	int doParallel = 1;
	int calcFluxes = 1;
	
	int intSteps = 0;  // Number of network integration steps taken
		
	// Compute the temperature-dependent factors for the rates.  Since we assume the GPU integration
	// to be done at constant temperature and density, these only need be calculated once per GPU call.
	
	float T93 = __powf(T9, THIRD);
	float t1 = 1/T9;
	float t2 = 1/T93;
	float t3 = T93;
    float t4 = T9;
    float t5 = T93*T93*T93*T93*T93;
    float t6 = __logf(T9);
	
	int i = threadID();    // Unique thread index for arbitrary number of blocks
	
	if(doParallel == 1)
	{
		// Parallel version: Compute all rates in parallel, with CUDA supplying a separate thread for 
		// each value of i.
		
		if(i < numberReactions)    // Prevent processing on threads outside bounds of network
		Rate[i] =   
			Prefac[i]* __expf(P0[i] + t1*P1[i] + t2*P2[i] + t3*P3[i] + t4*P4[i] + t5*P5[i] + t6*P6[i]); 
		
	} else {
	
		// Serial version for reference
		
		for(int i=0; i<numberReactions; i++)
		{
			Rate[i] =
			  Prefac[i]* __expf(P0[i] + t1*P1[i] + t2*P2[i] + t3*P3[i] + t4*P4[i] + t5*P5[i] + t6*P6[i]); 
		}
		
	}
	
	__syncthreads();
		
	/*
	 * Begin the time integration from t=0 to tmax. Rather than t=0 we start at some very
	 * small value of t.
	 */
	
	float t = 1.0e-16;             // The current integration time
	float dt = dt_init;            // The current integration timestep
	float prevdt = dt_init;        // The integration timestep from the previous step
		
	// Main time integration loop
	
	while(t < tmax)
	{		
		// Compute the fluxes from the previously-computed rates and the current abundances
		
		if(calcFluxes == 1)
		{
					
			if(doParallel == 1)
			{
				// Parallel version 
				
				if(i < numberReactions)

				{
					
					int nr = NumReactingSpecies[i];        // Number reacting species (1, 2, or 3)
					
					__syncthreads();

					// Switch on whether 1-body, 2-body, or 3-body reaction
					
					
					switch(nr)
					{
						
						case 1:    // 1-body; flux = rate x Y

							Flux[i] = Rate[i] * Y[*(Reactant1+i)]; 
							
						break;
							
						case 2:    // 2-body; flux = rate x Y x Y
							
							Flux[i] = Rate[i] * Y[*(Reactant1+i)] * Y[*(Reactant2+i)]; 
							
						break;
							
						case 3:    // 3-body; flux = rate x Y x Y x Y
							
							Flux[i] = Rate[i] * Y[*(Reactant1+i)] * Y[*(Reactant2+i)] 
									* Y[*(Reactant3+i)]; 
							
						break;
					}
				}
							
			} else {
		
				// Serial version for reference
			
				for(int j=0; j<numberReactions; j++)
				{
					int nr = NumReactingSpecies[j];
					
					switch(nr)
					{		
						case 1:
							Flux[j] = Rate[j] * Y[*(Reactant1+j)];			
						break;
							
						case 2:					
							Flux[j] = Rate[j] * Y[*(Reactant1+j)] * Y[*(Reactant2+j)]; 				
						break;
							
						case 3:					
							Flux[j] = Rate[j] * Y[*(Reactant1+j)] * Y[*(Reactant2+j)] 
										* Y[*(Reactant3+j)];				
						break;
					}		
				}	
			} 
		}
		
		__syncthreads();
		
		// Populate the F+ and F- arrays in parallel from the master Flux array
		
		if(i < totalFplus) 
		{
			int indy = MapFplus[i];
			Fplus[i] = FplusFac[i]*Flux[indy];
		}
		__syncthreads();
		
		 		
		if(i < totalFminus) 
		{
			Fminus[i] = FminusFac[i]*Flux[MapFminus[i]];
		}
		__syncthreads();
		
		
		// Sum the F+ and F- for each isotope.  The outer loop (in i) is parallel
		// but the inner loops (in j) are serial.  Can we do better? Seems we should
		// be able to?
			
		if(i < numberSpecies)
		{		
			// Partially serial Sum F+
			int minny = 0;
			if(i>0) minny = FplusMax[i-1]+1;
			FplusSum[i] = 0.0f;	
			for(int j=minny; j<=FplusMax[i]; j++)
			{
				FplusSum[i] += Fplus[j];
			}
				
			// Partially serial Sum F-
			minny = 0;
			if(i>0) minny = FminusMax[i-1]+1;
			FminusSum[i] = 0.0f;
			for(int j=minny; j<=FminusMax[i]; j++)
			{
				FminusSum[i] += Fminus[j];
			}
		}

		__syncthreads();
		
			
		/*
		Now use the fluxes to update the populations in parallel for this timestep
		For now we shall assume the asymptotic method. We determine whether each isotope 
		satisfies the asymptotic condition. If it does we update with the asymptotic formula. 
		If not, we update numerically using the forward Euler formula.
		*/
		
		if(i < numberSpecies)
		{		
			if(checkAsy(Fminus[i], Y[i], dt) == 1)
			{
				Y[i] = asymptoticUpdate(FplusSum[i], FminusSum[i], Y[i], dt);
			}
			else
			{
				Y[i] += eulerUpdate(FplusSum[i], FminusSum[i], dt);
			}		
		}
		__syncthreads();
		
		// Increment the integration time and set the new timestep	
		
		t += dt;
		intSteps ++;
		prevdt = dt;
		dt = computeTimestep(prevdt, t, tmax);
		
		// Temporary diagnostic halt
		if(intSteps >= 99) break;
		
	}      //+++ End of time integration while-loop +++//
	
}      // +++ End of kernel integrateNetwork +++ //


// Function to determine whether an isotope specified by speciesIndex satisfies the
// asymptotic condition. Returns 1 if it does and 0 if not.

__device__ int checkAsy(float Fminus, float Y, float dt)
{
	if(Y>0.0f && Fminus*dt/Y > 1.0f)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}


// Function to return the updated Y using the asymptotic formula

__device__ float asymptoticUpdate(float Fplus, float Fminus, float Y, float dt)
{
	return (Y + Fplus*dt)/(1.0f + Fminus*dt/Y);  // Sophia He formula
}


// Function to return the Y specified by speciesIndex updated using the forward Euler method

__device__ float eulerUpdate(float FplusSum, float FminusSum, float dt)
{
	return (FplusSum-FminusSum)*dt;
}


/*
 * Construct a unique thread ID from the built-in CUDA variables threadIdx.x, blockIdx.x, and blockDim.x.
 * threadIdx.x is a unique thread ID within a block, blockIdx.x identifies the block, and blockDim.x 
 * is the number of threads in each block. (We also have available from CUDA gridDim.x, which is the 
 * number of threads in each block, but don't need it here.)  Thus the following definition ensures that the 
 * ID i is unique for every thread distributed on the device.  Note:  if all threads were assigned to a
 * single block, i=threadIdx.x would be a unique identifier.  But on a CUDA 1.1 device the maximum
 * number of threads per block is 512.  Thus, if the task requires more than 512 threads it must be
 * assigned to more than one block and i=threadIdx.x is no longer unique.
*/
	
__device__ int threadID()
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	return i;      // Return unique thread ID
}

// Function to compute the network timestep. For now it is a placeholder
// just returning the timestep as a fixed fraction of the time.

__device__ float computeTimestep(float prevdt, float t, float tmax)
{	
	float dt;
	if(t == 0.0f)
	{
		dt = 1.0e-20;
	}
	else
	{
		dt = 0.1f*t;
	}
	
	// Prevent final integration step from overstepping tmax
	//if(t+dt > tmax) dt = tmax - t;
	
	return dt;
}

// Function to compute the effective decay constant keff for asymptotic approximation
// *** NOT PRESENTLY USED ***

__device__ float computekeff(float Fminus, float Y)
{
	if(Y > 0)
	{
		return Fminus/Y;
	}
	else
	{
		return 0.0f;
	}
}



//---------- Code below is executed on the host (CPU) -----------//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>

// SIZE defines the number of reactions to be calculated.  Need min of SIZE=4395 for 365-element network, 
// 1603 for 150-isotope network, 2234 for the 194-isotope network, 3176 for the 268-isotope network,
// 48 for the alpha network, and 1566 for the nova134 network. These sizes are hardwired for now but 
// eventually we want to assign them dynamically.

#define SIZE 48                         // Max number of reactions
#define ISOTOPES 16                      // Max isotopes in network (16 for alpha network)
#define LABELSIZE 35                      // Max size of reaction string a+b>c in characters
#define PF 24                             // # entries in partition function table for each isotope
#define THIRD 0.333333333333333


// Define some CPU timing utilities. Tends to return zero for anything that takes
// less than about 10 ms.  Usage:
//     START_CPU;
//     ... code to be timed ...
//     STOP_CPU;
//     PRINT_CPU
// in the host code. These may be used to time CPU processes in the host code.

clock_t startCPU, stopCPU;
#define START_CPU if ((startCPU=clock())==-1) {printf("Error calling clock"); exit(1);}
#define STOP_CPU if ((stopCPU=clock())==-1) {printf("Error calling clock"); exit(1);}
#define PRINT_CPU (printf("\nTimer: %f ms used by CPU\n",1000*(double)(stopCPU-startCPU)/CLOCKS_PER_SEC));


// Define some GPU timing utilities. These are invoked from the host program. Usage:
//     START_GPU;
//         kernelFunction <<< numBlocks, threadsPerBlock >>> (args)
//     STOP_GPU;
//     PRINT_GPU
// in the host code. This estimates the time for the kernel kernelFunction to run on the GPU.
// For a more extensive discusion, see Section 5.1.2 of the CUDA Best Practices Guide at
// http://developer.download.nvidia.com/compute/DevZone/docs/html/C/doc/CUDA_C_Best_Practices_Guide.pdf

float timeGPU;
hipEvent_t start, stop;
#define START_GPU hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define STOP_GPU hipEventRecord(stop, 0); hipEventSynchronize(stop);\
   hipEventElapsedTime(&timeGPU, start, stop);\
   hipEventDestroy(start);hipEventDestroy(stop);
#define PRINT_GPU printf("\n\nTime to compute on GPU: %f ms \n", timeGPU);
   
// Define a utility to check for CUDA errors.  Place it immediately after a CUDA kernel
// call in the host code. The initial cudaDeviceSynchronize() command ensures that the device
// has completed all preceding requested tasks.

#define CUDA_ERROR_CHECK hipDeviceSynchronize(); hipError_t error = hipGetLastError();\
   if(error != hipSuccess){printf("***CUDA error: %s\n", hipGetErrorString(error)); exit(-1);}\
   else{printf("\nNo CUDA errors detected\n" );}

FILE *fr;            // File pointer for data read-in

// Variables to hold data read in

float P0[SIZE];       // Array holding library rate parameters p0
float P1[SIZE];       // Array holding library rate parameters p1
float P2[SIZE];       // Array holding library rate parameters p2
float P3[SIZE];       // Array holding library rate parameters p3
float P4[SIZE];       // Array holding library rate parameters p4
float P5[SIZE];       // Array holding library rate parameters p5
float P6[SIZE];       // Array holding library rate parameters p6
float Q[SIZE];        // Array holding library entry for reaction Q-value
float Rate[SIZE];     // Array holding the computed rate for each reaction
float Flux[SIZE];     // Array holding the computed fluxes for each reaction

int reaclibClass[SIZE];         // Reaclib class index for reaction (1-8) 
int RGclass[SIZE];              // Reaction Group class (PE) for reaction (1-5)
int RGmemberIndex[SIZE];        // Member index within reaction group
int NumReactingSpecies[SIZE];         // Number of reactant isotopes for the reaction
int NumProducts[SIZE];          // Number of product isotopes for the reaction
int isEC[SIZE];                 // Whether reaction is electron capture (0=false; 1=true)
int isReverseR[SIZE];           // Whether reaction is inverse (matters for partition func)
float Prefac[SIZE];             // The statistical prefactor for each reaction

int reactantZ[SIZE][4];          // Holds Z for each reactant isotope
int reactantN[SIZE][4];          // Holds N for each reactant isotope
int productZ[SIZE][4];           // Holds Z for each product isotope
int productN[SIZE][4];           // Holds N for each product isotope
int ReactantIndex[SIZE][4];      // Index of isotope vector for each reactant isotope
int ProductIndex[SIZE][4];       // Index of isotope vector for each product isotope
int Reactant1[SIZE];          // Isotope index of first reactant
int Reactant2[SIZE];          // Isotope index of second reactant (if 2-body or 3-body)
int Reactant3[SIZE];		 // Isotope index of third reactant (if 3-body)

int Z[ISOTOPES];                // Array holding Z values for isotopes
int N[ISOTOPES];                // Array holding N values for isotopes
float AA[ISOTOPES];             // Array holding A values for isotopes
float Y[ISOTOPES];              // Array holding abundances Y for isotopes
float Diagnose[DIAGNOSE_SIZE];  // Diagnostics array
float X[ISOTOPES];              // Array holding mass fractions X for isotopes
float massExcess[ISOTOPES];     // Array holding mass excesses for isotopes
char isoLabel[ISOTOPES][5];     // Isotope labels

int numberSpecies;              // Total number of species in network
int numberReactions;            // Total number of reactions in the network

char reacLabel[SIZE][LABELSIZE];   // Character array holding reaction labels

// Temperatures in units of 10^9 K for partition function table (see pf[][]). These are
// copied from the corresponding variable Tpf[] in the Java code.

const float Tpf[] = { 0.1f, 0.15f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f, 0.9f, 1.0f,
	1.5f, 2.0f, 2.5f, 3.0f, 3.5f, 4.0f, 4.5f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f };
	
// Array holding partition function values for the 24 temperatures given in Tpf[]
// for each isotope
	
float pf[ISOTOPES][PF]; 

float Dens[3];

// Array with entries +1 if a reaction increases the population of the isotope (contributes to 
// F+), -1 if it decreases it (contributes to F-) and 0 if the reaction does not change the population
// of the isotope. This array is populated in the function parseF().  It is characteristic of the
// structure of the network and thus has to be calculated only once for a given network.

int reacMask[ISOTOPES][SIZE];

// Function Signatures:
void devcheck(int);
void readLibraryParams(char *);
void readNetwork(char *);
void writeNetwork(void);
void testTimerCPU(void);
void computeRatesCPU(float);
void writeRates(char *);
void writeAbundances(void);
void parseF(void);

// Filename for input rates library data. The file rateLibrary.data output by the Java code through 
// the stream toRateData has the expected format for this file. Standard test cases: 
// rateLibrary_alpha.data, rateLibrary_150.data, rateLibrary_365.data, rateLibrary_nova134.data.

char rateLibraryFile[] = "rateLibrary_alpha.data";

// Filename for network + partition function input.  The file output/CUDAnet.inp
// output by the Java code through the stream toCUDAnet has the expected format for 
// this file. Standard test cases: CUDAnet_alphasolar.inp, CUDAnet_150solar.inp,
// CUDAnet_365solar.inp, CUDAnet_nova134.inp.

char networkFile[] = "CUDAnet_alpha.inp";

// Control some of the printout (1 to include, 0 to suppress)
int displayInput = 0;
int ratesCPU = 0;

// Total number of F+ and F- terms in the network
int totalFplus = 0;
int totalFminus = 0;

// Arrays to hold non-zero fluxes in the network. These will be allocated dynamically 
// below with malloc

float* Fplus;        // Dynamically-allocated 1D array for non-zero F+ (Dim totalFplus)
float* Fminus;       // Dynamically-allocated 1D array for non-zero F- (Dim totalFminus)

// Arrays to hold number species factors for F+ and F- arrays. For example, for 12C+12C ->
// 4He + 20Ne the species factor is 1 for 4He and 20Ne diff. equation terms but 2 for
// 12C diff equation term (since the reaction involves one 4He and one 20Ne, but two 12C.

float* FplusFac;     // Dynamically allocated 1D array for number species factor in F+ terms
float* FminusFac;    // Dynamically allocated 1D array for number species factor in F- terms

float* FplusSum;     // Sum of F+ for each isotope
float* FminusSum;    // Sum of F- for each isotope

int* FplusMax;       // Upper index for each isotope in the Fplus array
int* FplusMin;       // Lower index for each isotope in the Fplus array
int* FminusMax;      // Upper index for each isotope in the Fminus array
int* FminusMin;      // Lower index for each isotope in the Fminus array

// Arrays to hold index of upper boundary for entries of each isotope in the Fplus and
// Fminus arrays. These will be allocated dynamically below malloc

int* FplusIsotopeCut;    // Upper index for each isotope in Fplus (Dim numberSpecies)
int* FminusIsotopeCut;   // Upper index for each isotope in Fminus (Dim numberSpecies)

int* numFluxPlus;        // Number of finite F+ components for each isotope (Dim numberSpecies)
int* numFluxMinus;       // Number of finite F- components for each isotope (Dim numberSpecies)

int* FplusIsotopeIndex;  // Array containing the isotope index for each F+ (Dim totalFplus)
int* FminusIsotopeIndex; // Array containing the isotope index for each F- (Dim totalFminus)

/*
 * Arrays to hold the mapping of the Fplus and Fminus arrays of fluxes to the
 * master flux array Flux[SIZE]. For example, MapFplus[0] will hold the value of
 * the index i in Flux[i] that corresponds to the reaction that generates Fplus[0].
 * These will be allocated dynamically below.
*/

int* MapFplus;    // Index mapper for Fplus (Dim totalFplus)
int* MapFminus;   // Index mapper for Fminus (Dim totalFminus)

// Arrays for temporary storage. Will be allocated dynamically below
int* tempInt1;
int* tempInt2;

// Control diagnostic printout (1 to print, 0 to suppress)
int showParsing = 0;
int showFparsing = 0;

// Array of integer input parameters.  Needed because we can't pass
// too many arguments to the kernel (256 byte limit for 1.1 devices).

int Params1[4];

// Array of float input parameters.  Needed because we can't pass
// too many arguments to the kernel (256 byte limit for 1.1 devices).

float Params2[3];


// Main CPU routine

int main()
{
	// Ensure that a valid device (GPU) exists 
	printf("\nChecking for valid device:\n");
	devcheck(0);
	
	// Check available memory on the GPU	
	size_t msizeFree;
	size_t msizeTotal;	
	hipMemGetInfo(&msizeFree, &msizeTotal);	 
	printf("\nGPU total memory: %d\nGPU free memory: %d", (int)msizeTotal, (int)msizeFree);
	
	// Following memory queries not supported on GF 8600 GT	
	//  size_t msize;
	// 	cudaDeviceGetLimit(&msize, cudaLimitMallocHeapSize);
	//  printf("\nGPU heap size: %d", (int)msize);
	// 	cudaDeviceGetLimit(&msize, cudaLimitStackSize);
	// 	printf("\nGPU stack size for each thread: %d\n", (int)msize);
	
	// Set the temperature in units of 10^9 K and density in units of g/cm^3. The 
	// temperature and density will be passed from the hydro code in an operator-split 
	// coupling of this network to hydro. These will be used to calculate the reaction
	// rates in the network on the GPU. Since we are assuming operator splitting, the
	// temperature and density are assumed constant for the entire network integration
	// on the gPU.
	
	float T9 = 6.0f;
	float rho = 1.0e8;
	
	// Set the range of time integration and the initial timestep.  In an operator-split
	// coupling tmax will come from the hydro and dt_init will likely be the last timestep
	// of the previous network integration (for the preceding hydro timestep).
	
	float tmax = 1e-11;
	float dt_init = 1e-17;               
	
	// Read in rate library data from a file. This is required only once, at the
	// beginning of the entire calculation.
	
	char *rateLibraryFilePtr = rateLibraryFile;
	readLibraryParams(rateLibraryFilePtr);
	
	// Multiply the prefactor by the appropriate density factors (1 for 1-body,
	// rho for 2-body, and rho^2 for 3-body. This is required at the beginning of
	// each network integration of the hydro timestep, since the density will generally
	// change over a hydro timestep in each zone.
	
	Dens[0] = 1.0f;
	Dens[1] = rho;
	Dens[2] = rho*rho;
	
	for(int i=0; i<SIZE; i++)
	{
		Prefac[i] *= (Dens[NumReactingSpecies[i]-1]);
	}
	
	// Optionally compute the temperature-dependent rates first on the CPU
	if(ratesCPU == 1){
		
		// Test the CPU timer by executing a long, pointless loop
		testTimerCPU();
		
		START_CPU     // Start a timer for the actual calculation
		
		// First compute the rates serially for reference using the CPU
			
		computeRatesCPU(T9);
		
		STOP_CPU;     // Stop the timer
		PRINT_CPU;    // Print timing information
		
		// Display the rates. (Note the two different techniques used in calling
		// writeRates here and for "GPU" below). Here we use a pointer; in the
		// example below we pass an array.)
		
		char label[] = "on CPU";
		char *labelPtr = label;
		writeRates(labelPtr);
	
	}
	
	// Read in network file and associated partition functions.  This is required only
	// once at the very beginning of the calculation.
	
	char *networkFilePtr = networkFile;
	readNetwork(networkFilePtr);
	writeNetwork();

	// Find for each isotope all reactions that change its population.  This analysis of
	// the network is required only once at the very beginning of the calculation (provided
	// that the network species and reactions remain the same for the entire calculation).
	// The work is done by the function parseF().
	
	// Number of F+ and F- components for each isotope
	numFluxPlus = (int*) malloc(sizeof(int) * numberSpecies);
	numFluxMinus = (int*) malloc(sizeof(int) * numberSpecies);
	
	// Arrays for temporary storage
	tempInt1 = (int*) malloc(sizeof(int) * numberSpecies * numberReactions/2);
	tempInt2 = (int*) malloc(sizeof(int) * numberSpecies * numberReactions/2);
	
	parseF();
	
	// Create 1D arrays to hold non-zero F+ and F- for all reactions for all isotopes,
	// the arrays holding the species factors FplusFac and FminusFac, 
	// and also arrays to hold their sums for each isotope. Note that parseF() must
	// be run first because it determines totalFplus and totalFminus.
	
	Fplus = (float*) malloc(sizeof(float) * totalFplus);
	Fminus = (float*) malloc(sizeof(float) * totalFminus);
	FplusFac = (float*) malloc(sizeof(float) *totalFplus);
	FminusFac = (float*) malloc(sizeof(float) * totalFminus);
	FplusSum = (float*) malloc(sizeof(float) * numberSpecies);
	FminusSum = (float*) malloc(sizeof(float) * numberSpecies);
	
	// Arrays that hold the index of the boundary between different isotopes in the
	// Fplus and Fminus 1D arrays. Since FplusMax and FplusMin are related, and likewise
	// FminusMax and FminusMin are related, we will only need to pass FplusMax and
	// FminusMax to the kernel.
	
	FplusMax = (int*) malloc(sizeof(int) * numberSpecies);
	FplusMin = (int*) malloc(sizeof(int) * numberSpecies);
	FminusMax = (int*) malloc(sizeof(int) * numberSpecies);
	FminusMin = (int*) malloc(sizeof(int) * numberSpecies);
	
	// Fill the integer parameter array to pass to the kernel. Doing this to bypass
	// the formal limit of 256 for arguments passed to the kernel.
	
	Params1[0] = numberSpecies;
	Params1[1] = numberReactions;
	Params1[2] = totalFplus;
	Params1[3] = totalFminus;
	
	// Fill the float parameter array to pass to the kernel. Doing this to bypass
	// the formal limit of 256 for arguments passed to the kernel.
	
	Params2[0] = T9;
	Params2[1] = tmax;
	Params2[2] = dt_init; 		

	// Create 1D arrays that will be used to map finite F+ and F- to the Flux array.
	
	FplusIsotopeCut = (int*) malloc(sizeof(int) * numberSpecies);
	FminusIsotopeCut = (int*) malloc(sizeof(int) * numberSpecies);
	
	FplusIsotopeIndex = (int*) malloc(sizeof(int) * totalFplus);
	FminusIsotopeIndex = (int*) malloc(sizeof(int) * totalFminus);
	
	// Create 1D arrays that will hold the index of the isotope for the F+ or F- term
	MapFplus = (int*) malloc(sizeof(int) * totalFplus);
	MapFminus = (int*) malloc(sizeof(int) * totalFminus);
	
	FplusIsotopeCut[0] = numFluxPlus[0];
	FminusIsotopeCut[0] = numFluxMinus[0];
	for(int i=1; i<numberSpecies; i++)
	{
		FplusIsotopeCut[i] = numFluxPlus[i] + FplusIsotopeCut[i-1];
		FminusIsotopeCut[i] = numFluxMinus[i] + FminusIsotopeCut[i-1];
	}
	
	int currentIso = 0;
	for(int i=0; i<totalFplus; i++)
	{
		FplusIsotopeIndex[i] = currentIso;
		if(i == (FplusIsotopeCut[currentIso]-1)) currentIso ++;
	}
	
	currentIso = 0;
	for(int i=0; i<totalFminus; i++)
	{
		FminusIsotopeIndex[i] = currentIso;
		if(i == (FminusIsotopeCut[currentIso]-1)) currentIso ++;
	}
	
	// Diagnostic output
	if(showFparsing == 1)
	{
		printf("\n\n\nMAX F+ and F- INDEX FOR EACH ISOTOPE:\n");	
		for(int i=0; i<numberSpecies; i++)
		{
			printf("\nIsotope index = %d  %s  Max index F+ = %d  Max index F- = %d", 
				i, isoLabel[i], FplusIsotopeCut[i]-1, FminusIsotopeCut[i]-1);
		}
	}
				
	for(int i=0; i<totalFplus; i++)
	{
		MapFplus[i] = tempInt1[i];
	}
	
	for(int i=0; i<totalFminus; i++)
	{
		MapFminus[i] = tempInt2[i];
	}
	
	// Populate the FplusMin and FplusMax arrays
	FplusMin[0] = 0;
	FplusMax[0] = numFluxPlus[0]-1;
	for(int i=1; i<numberSpecies; i++)
	{
		FplusMin[i] = FplusMax[i-1] + 1;
		FplusMax[i] = FplusMin[i] + numFluxPlus[i] -1 ;	
	}
	// Populate the FminusMin and FminusMax arrays
	FminusMin[0] = 0;
	FminusMax[0] = numFluxMinus[0]-1;
	for(int i=1; i<numberSpecies; i++)
	{
		FminusMin[i] = FminusMax[i-1] + 1;
		FminusMax[i] = FminusMin[i] + numFluxMinus[i] -1 ;	
	}
	
	// Populate the FplusFac and FminusFac arrays that hold the factors counting the
	// number of occurences of the species in the reaction.  Note that this can only
	// be done after parseF() has been run to give reacMask[i][j].
	
	int tempCountPlus = 0;
	int tempCountMinus = 0;
	for(int i=0; i<ISOTOPES; i++)
	{
		for(int j=0; j<SIZE; j++)
		{
			if(reacMask[i][j] > 0)
			{
				FplusFac[tempCountPlus] = (float)reacMask[i][j];
// 				printf("\n F+  tempCountPlus=%d i=%d j=%d FplusFac=%3.1f", tempCountPlus, 
// 					   i, j, FplusFac[tempCountPlus]);
				tempCountPlus ++;
			}
			else if(reacMask[i][j] < 0)
			{
				FminusFac[tempCountMinus] = -(float) reacMask[i][j];
// 				printf("\n F-  tempCountMinus=%d i=%d j=%d FminusFac=%3.1f", tempCountMinus, 
// 					   i, j, FminusFac[tempCountMinus]);
				tempCountMinus ++;
			}	
		}
	}
	
	// Diagnostic output
	
	if(showFparsing == 1)
	{
		printf("\n\n\n---------- %d NON-VANISHING F+ SOURCE TERMS ----------\n", totalFplus);
		printf("\ndY[%s]/dt = dY[%d]/dt F+ source terms (%d):", 
					isoLabel[FplusIsotopeIndex[0]], FplusIsotopeIndex[0],
					numFluxPlus[FplusIsotopeIndex[0]]);
		for(int i=0; i<totalFplus; i++)
		{
			printf("\n   Isotope index = %d F+ index = %d Reac index = %d  %s", 
					FplusIsotopeIndex[i], i,
				MapFplus[i], reacLabel[MapFplus[i]]); 
			if(i == (FplusIsotopeCut[FplusIsotopeIndex[i]] - 1)  && i != totalFplus-1)
			{
				printf("\n");
				printf("\ndY[%s]/dt = dY[%d]/dt F+ source terms (%d):", 
						isoLabel[FplusIsotopeIndex[i+1]], FplusIsotopeIndex[i+1],
						numFluxPlus[FplusIsotopeIndex[i+1]]);
			}
		}	
		
		printf("\n\n\n---------- %d NON-VANISHING F- SOURCE TERMS ----------\n", totalFminus);
		printf("\ndY[%s]/dt = dY[%d]/dt F- source terms (%d):", 
					isoLabel[FminusIsotopeIndex[0]], FminusIsotopeIndex[0],
					numFluxMinus[FminusIsotopeIndex[0]]
			);
		for(int i=0; i<totalFminus; i++)
		{
			printf("\n   Isotope index = %d F- index = %d Reac index=%d  %s", 
					FminusIsotopeIndex[i], i, MapFminus[i], reacLabel[MapFminus[i]]);
			if(i == (FminusIsotopeCut[FminusIsotopeIndex[i]] - 1) && i != totalFminus-1 )
			{
				printf("\n");
				printf("\ndY[%s]/dt = dY[%d]/dt F- source terms (%d):", 
					isoLabel[FminusIsotopeIndex[i+1]], FminusIsotopeIndex[i+1],
					numFluxMinus[FminusIsotopeIndex[i+1]]
					);
			}
		}
		
		printf("\n");
	}
	
	// Set up the device pointers corresponding to the arrays.  Required only once at the
	// very beginning of the calculation.
	
	float *devPtrP0;
	float *devPtrP1; 
	float *devPtrP2; 
	float *devPtrP3;
	float *devPtrP4;
	float *devPtrP5;
	float *devPtrP6;
	float *devPtrPrefac;
	float *devPtrQ;
	float *devPtrRate;
    float *devPtrFlux;
	float *devPtrFplus;
	float *devPtrFminus;
	float *devPtrFplusFac;
	float *devPtrFminusFac;
	float *devPtrFplusSum;
	float *devPtrFminusSum;
	int *devPtrFplusMax;
	int *devPtrFminusMax;
	int *devPtrMapFplus;
	int *devPtrMapFminus;
	float *devPtrY;
	float *devPtrDiagnose;
	int *devPtrZ;                       // Z is not presently used in the kernel
	int *devPtrN;                       // N is not presently used in the kernel
	int *devPtrParams1;
	float *devPtrParams2;
	int *devPtrNumReactingSpecies;
    int *devPtrReactant1;
	int *devPtrReactant2;
	int *devPtrReactant3;
	
	// Allocate float and int memory on the device (the GPU) for all variables.  Required only
	// once at the very beginning of the calculation.
	
	int memsize1 = SIZE*sizeof(float);         // Memory size for floats labeled by reaction index
	int memsize2 = SIZE*sizeof(int);           // Memory size for ints labeled by reaction index
	int memsize3 = ISOTOPES*sizeof(float);     // Memory size for floats labeled by isotope index
	int memsize4 = ISOTOPES*sizeof(int);       // Memory size for ints labeled by isotope index
	int memsize5 = totalFplus*sizeof(float);   // Memory size for contributing F+ values
	int memsize6 = totalFminus*sizeof(float);  // Memory size for contributing F- values
	int memsize7 = totalFplus*sizeof(int);     // Memory size for contributing F+ value indices
	int memsize8 = totalFminus*sizeof(int);    // Memory size for contributing F- value indices
	
	hipMalloc((void**)&devPtrP0, memsize1); 
	hipMalloc((void**)&devPtrP1, memsize1); 
	hipMalloc((void**)&devPtrP2, memsize1);
	hipMalloc((void**)&devPtrP3, memsize1);
	hipMalloc((void**)&devPtrP4, memsize1);
	hipMalloc((void**)&devPtrP5, memsize1);
	hipMalloc((void**)&devPtrP6, memsize1);
	hipMalloc((void**)&devPtrPrefac, memsize1);
	hipMalloc((void**)&devPtrQ, memsize1);
	hipMalloc((void**)&devPtrRate, memsize1);
    hipMalloc((void**)&devPtrFlux, memsize1);
	hipMalloc((void**)&devPtrFplus, memsize5);
	hipMalloc((void**)&devPtrFminus, memsize6);
	hipMalloc((void**)&devPtrFplusFac, memsize5);
	hipMalloc((void**)&devPtrFminusFac, memsize6);
	hipMalloc((void**)&devPtrFplusSum, memsize3);
	hipMalloc((void**)&devPtrFminusSum, memsize3);
	hipMalloc((void**)&devPtrFplusMax, memsize4);
	hipMalloc((void**)&devPtrFminusMax, memsize4);	
	hipMalloc((void**)&devPtrMapFplus, memsize7);
	hipMalloc((void**)&devPtrMapFminus, memsize8);
	hipMalloc((void**)&devPtrY, memsize3);
	hipMalloc((void**)&devPtrDiagnose, DIAGNOSE_SIZE*sizeof(float));
	hipMalloc((void**)&devPtrZ, memsize4);
	hipMalloc((void**)&devPtrN, memsize4);
	hipMalloc((void**)&devPtrParams1, 4*sizeof(int));
	hipMalloc((void**)&devPtrParams2, 3*sizeof(float));
	hipMalloc((void**)&devPtrNumReactingSpecies, memsize2);
    hipMalloc((void**)&devPtrReactant1, memsize2);
	hipMalloc((void**)&devPtrReactant2, memsize2);
	hipMalloc((void**)&devPtrReactant3, memsize2);
	
	/*
	 * Copy array memory to the device using cudaMemcpy(void* destination, 
	 * const void* source, size_t memSize, enum cudaMemcpyKind kind), which 
	 * copies memSize bytes from the memory area pointed to by source to the 
	 * memory area pointed to by destination, with kind=cudaMemcpyHostToDevice 
	 * or kind=cudaMemcpyDeviceToHost, or kind=cudaMemcpyDeviceToDevice 
	 * specifying the nature (direction) of the copy. Note: in C an array name
	 * is effectively a pointer.  This is required only once at the very beginning
	 * of the calculation. These quantities will then reside on the GPU for the
	 * duration of the calculation (over all hydro timesteps).
	*/
	
	hipMemcpy(devPtrP0, P0, memsize1, hipMemcpyHostToDevice); 
	hipMemcpy(devPtrP1, P1, memsize1, hipMemcpyHostToDevice); 
	hipMemcpy(devPtrP2, P2, memsize1, hipMemcpyHostToDevice);
	hipMemcpy(devPtrP3, P3, memsize1, hipMemcpyHostToDevice); 
	hipMemcpy(devPtrP4, P4, memsize1, hipMemcpyHostToDevice); 
	hipMemcpy(devPtrP5, P5, memsize1, hipMemcpyHostToDevice); 
	hipMemcpy(devPtrP6, P6, memsize1, hipMemcpyHostToDevice);
	hipMemcpy(devPtrPrefac, Prefac, memsize1, hipMemcpyHostToDevice);
	hipMemcpy(devPtrQ, Q, memsize1, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFplus, Fplus, memsize5, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFminus, Fminus, memsize6, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFplusFac, FplusFac, memsize5, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFminusFac, FminusFac, memsize6, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFplusSum, FplusSum, memsize3, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFminusSum, FminusSum, memsize3, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFplusMax, FplusMax, memsize4, hipMemcpyHostToDevice);
	hipMemcpy(devPtrFminusMax, FminusMax, memsize4, hipMemcpyHostToDevice);
	hipMemcpy(devPtrMapFplus, MapFplus, memsize7, hipMemcpyHostToDevice);
	hipMemcpy(devPtrMapFminus, MapFminus, memsize8, hipMemcpyHostToDevice);
	hipMemcpy(devPtrY, Y, memsize3, hipMemcpyHostToDevice);
	hipMemcpy(devPtrDiagnose, Diagnose, DIAGNOSE_SIZE*sizeof(float), hipMemcpyHostToDevice);  // Needed?
	hipMemcpy(devPtrZ, Z, memsize4, hipMemcpyHostToDevice);
	hipMemcpy(devPtrN, N, memsize4, hipMemcpyHostToDevice);
	hipMemcpy(devPtrParams1, Params1, 4*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devPtrParams2, Params2, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devPtrNumReactingSpecies, NumReactingSpecies, memsize2, hipMemcpyHostToDevice);
    hipMemcpy(devPtrReactant1, Reactant1, memsize2, hipMemcpyHostToDevice);
	hipMemcpy(devPtrReactant2, Reactant2, memsize2, hipMemcpyHostToDevice);
	hipMemcpy(devPtrReactant3, Reactant3, memsize2, hipMemcpyHostToDevice);

	/*
	 * Define the number of blocks on the grid and the number of
	 * threads per block. The number of reactions is given by SIZE, so
	 * the product of numBlocks.x and threadsPerBlock.x must be greater
	 * than or equal to SIZE. If we choose threadsPerBlock.x = 256,
	 * we require at least numBlocks.x = 1 for the alpha network, 7 for the 150-
	 * isotope network, 9 for the 194-isotope network, 13 for the 268-
	 * isotope network, 18 for the 365-isotope network, and 7 for the
	 * nova134 network. Being done by hand now, but eventually we should 
	 * automate this. Required only once at the very beginning of the calculation.
	*/
	
	
	dim3 numBlocks(4, 1, 1);
	dim3 threadsPerBlock(BLOCKSIZE, 1, 1);
	
	/* 
	 * Launch kernel integrateNetwork on GPU. This will execute the global function 
	 * integrateNetwork on the device using numBlocks blocks, each with threadsPerBlock 
	 * threads.  Pass pointers for the arrays and values for the scalars.  This kernel 
	 * will be launched once for every hydro timestep, with the entire network calculation 
	 * for the hydro timestep being done on the GPU within this kernel. At the beginning 
	 * we must pass to the kernel running the network on the GPU the current temperature, 
	 * the proposed initial network timestep (likely we will choose the timestep from the
	 * end of the network integration for the preceding hydro timestep), and the current 
	 * abundances Y[i] for the species in the network (which may have been altered in the 
	 * hydro timestep by processes like advection. 
	*/
	
	START_GPU;     // Start timer for device code
	
	integrateNetwork <<< numBlocks, threadsPerBlock >>> 
	(
		devPtrP0, 
		devPtrP1, 
		devPtrP2, 
		devPtrP3, 
		devPtrP4, 
		devPtrP5, 
		devPtrP6, 
		devPtrPrefac,  
		devPtrQ, 
		devPtrRate, 
		devPtrFlux, 
		devPtrFplus,
        devPtrFminus,
		devPtrFplusFac,
		devPtrFminusFac,
		devPtrFplusSum,
		devPtrFminusSum,
 		devPtrFplusMax,
 		devPtrFminusMax,
        devPtrMapFplus,
		devPtrMapFminus,
		devPtrY, 
		devPtrDiagnose,
		devPtrZ, 
		devPtrN,
		devPtrParams1,
		devPtrParams2,
        devPtrNumReactingSpecies,
		devPtrReactant1, 
		devPtrReactant2, 
		devPtrReactant3
	);

	STOP_GPU;           // Stop timer for device code
	PRINT_GPU;          // Print timing for device code
	
	CUDA_ERROR_CHECK    // Check for CUDA errors
	
	// The integration is finished on the GPU.  Copy the results from the device back 
	// to the host.
	
	hipMemcpy(Rate, devPtrRate, memsize1, hipMemcpyDeviceToHost);
    hipMemcpy(Flux, devPtrFlux, memsize1, hipMemcpyDeviceToHost);
	hipMemcpy(Fplus, devPtrFplus, memsize5, hipMemcpyDeviceToHost);
	hipMemcpy(Fminus, devPtrFminus, memsize6, hipMemcpyDeviceToHost);
	hipMemcpy(FplusFac, devPtrFplusFac, memsize5, hipMemcpyDeviceToHost);
	hipMemcpy(FminusFac, devPtrFminusFac, memsize6, hipMemcpyDeviceToHost);
	hipMemcpy(FplusSum, devPtrFplusSum, memsize3, hipMemcpyDeviceToHost);
	hipMemcpy(FminusSum, devPtrFminusSum, memsize3, hipMemcpyDeviceToHost);
	hipMemcpy(FplusMax, devPtrFplusMax, memsize4, hipMemcpyDeviceToHost);
	hipMemcpy(FminusMax, devPtrFminusMax, memsize4, hipMemcpyDeviceToHost);
	hipMemcpy(MapFplus, devPtrMapFplus, memsize7, hipMemcpyDeviceToHost);
	hipMemcpy(MapFminus, devPtrMapFminus, memsize8, hipMemcpyDeviceToHost);
	hipMemcpy(Y, devPtrY, memsize3, hipMemcpyDeviceToHost);
	hipMemcpy(Diagnose, devPtrDiagnose, DIAGNOSE_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	
	// Display the results returned from the GPU
    char label2[] = "on GPU";
    writeRates(label2);
	
	// Print the final values of F+ and F- transferred from GPU
	printf("\n\nFINAL F+ VALUES:\n");
	for(int i=0; i<totalFplus; i++)
	{
		printf("\nF+[%d] = %7.4e  Increases Y[%s] through %s  MapIndex=%d  FplusFac=%3.1f", 
			i, Fplus[i], isoLabel[FplusIsotopeIndex[i]], 
			reacLabel[MapFplus[i]], MapFplus[i], FplusFac[i]);
	}
	printf("\n\n\nFINAL F- VALUES:\n");
	for(int i=0; i<totalFminus; i++)
	{
		printf("\nF-[%d] = %7.4e  Decreases Y[%s] through %s  MapIndex=%d  FminusFac=%3.1f", 
			i, Fminus[i], isoLabel[FminusIsotopeIndex[i]], 
			reacLabel[MapFminus[i]], MapFminus[i], FminusFac[i]);
	}
		
	printf("\n\n\nF+ and F- MIN AND MAX FOR EACH ISOTOPE:\n");
	for(int i=0; i<numberSpecies; i++)
	{
		printf("\n%3d %5s F+min=%3d F+max=%3d F-min=%3d F-max=%d", 
			   i, isoLabel[i], FplusMin[i], FplusMax[i], FminusMin[i], FminusMax[i]);
	}
	
	printf("\n\n\nSUM OF FLUXES FOR EACH ISOTOPE:\n");
	
	// Print the total F+ and F- for each isotope transferred from the GPU
	float totalFplus = 0.0f;
	float totalFminus = 0.0f;
	for(int i=0; i<numberSpecies; i++)
	{
		printf("\n%3d %5s  sumF+=%10.4e  sumF-=%10.4e Fnet=%10.4e Y=%10.4e", 
			   i, isoLabel[i], FplusSum[i], FminusSum[i], FplusSum[i]-FminusSum[i], Y[i]);
		totalFplus += FplusSum[i];
		totalFminus += FminusSum[i];
	}
	
	printf("\n\ntotalF+ = %7.4e  totalF- = %7.4e", totalFplus, totalFminus);

	// Diagnostics returned from the GPU
	
	printf("\n\n\nDIAGNOSTICS:\n\n");
	for(int i=0; i<DIAGNOSE_SIZE; i++)
	{
		printf("Diagnose[%d]=%7.4e\n", i, Diagnose[i]);
	}
	
	printf("\n\nFINAL ABUNDANCES:\n");
	writeAbundances();
	
	PRINT_GPU;

	printf("\n");
	
    // Free memory allocated on the device  
	
	hipFree(devPtrP0); 
	hipFree(devPtrP1); 
	hipFree(devPtrP2);
	hipFree(devPtrP3);
	hipFree(devPtrP4);
	hipFree(devPtrP5);
	hipFree(devPtrP6);
	hipFree(devPtrPrefac);
	hipFree(devPtrQ);
	hipFree(devPtrRate);
    hipFree(devPtrFlux);
	hipFree(devPtrFplus);
	hipFree(devPtrFminus);
	hipFree(devPtrFplusFac);
	hipFree(devPtrFminusFac);
	hipFree(devPtrFplusSum);
	hipFree(devPtrFminusSum);
	hipFree(devPtrFplusMax);
	hipFree(devPtrFminusMax);
	hipFree(devPtrMapFplus);
	hipFree(devPtrMapFminus);
	hipFree(devPtrY);
	hipFree(devPtrDiagnose);
	hipFree(devPtrZ);
	hipFree(devPtrN);
	hipFree(devPtrParams1);
	hipFree(devPtrParams2);
	hipFree(devPtrNumReactingSpecies);
    hipFree(devPtrReactant1);
	hipFree(devPtrReactant2);
	hipFree(devPtrReactant3);
	
	// Free memory allocated on the CPU
	
	free(Fplus);
	free(Fminus);
	free(FplusFac);
	free(FminusFac);
	free(FplusSum);
	free(FminusSum);
	free(FplusMin);
	free(FplusMax);
	free(FminusMin);
	free(FminusMax);
	free(FplusIsotopeCut);
	free(FminusIsotopeCut);
	free(MapFplus);
	free(MapFminus);
	free(numFluxPlus);
	free(numFluxMinus);
	free(tempInt1);
	free(tempInt2);
	free(FplusIsotopeIndex);
	free(FminusIsotopeIndex);
	
}  // End main


// Function to compute the rates on the CPU rather than GPU.

void computeRatesCPU(float T9)
{	
	float T93 = powf(T9, THIRD); 
	float t1 = 1/T9;
	float t2 = 1/T93;
	float t3 = T93;
	float t4 = T9;
	float t5 = T93*T93*T93*T93*T93;
	float t6 = logf(T9);
	
	for (int i=0; i<numberReactions; i++){
		Rate[i] = 
			Prefac[i]*expf(P0[i] + t1*P1[i] + t2*P2[i] + t3*P3[i] + t4*P4[i] + t5*P5[i] + t6*P6[i]);
	}
}

// Function to print out all the rates. The label can be used to distinguish cases
// if called more than once.

void  writeRates(char *label)
{
	printf("\nCOMPUTED RATES (%s):\n\n", label);
	for (int i=0; i<numberReactions; i++) 
	{
		printf("%d %s Rate=%6.3e Y1=%6.3e Y2=%6.3e Y3=%6.3e Flux=%7.4e Q=%6.3f Prefac=%6.3e Reactants=%d\n",
			i,reacLabel[i], Rate[i], Y[Reactant1[i]], Y[Reactant2[i]], Y[Reactant3[i]], Flux[i], 
			Q[i], Prefac[i], NumReactingSpecies[i]);
	}
}

// Function to write out the abundances in the network

void writeAbundances()
{
	printf("\nIndex  Isotope   Abundance Y   Mass Frac X");
	float sumX = 0.0f;
	for(int i=0; i<ISOTOPES; i++)
	{
		float X = Y[i]*AA[i];
		sumX += X;
		printf("\n %4d     %4s    %8.4e    %8.4e", i, isoLabel[i], Y[i], X);
	}
	printf("\n\nsum X = %6.4f", sumX);
}

// Function to print out the network isotopes, mass excesses, and the entries in the 
// partition function table for each isotope.

void writeNetwork()
{
	printf("\n\n%d ISOTOPES IN NETWORK:\n\n",numberSpecies);
	printf("Index  Isotope   A   Z   N  Abundance Y  MassFrac X  MassXS(MeV)\n");
	for (int i=0; i<numberSpecies; i++) 
	{
		printf("%5d %8s %3d %3d %3d  %8.5e   %9.6f   %10.5f\n",  i, isoLabel[i], (int)AA[i], Z[i], N[i], 
			   Y[i], X[i], massExcess[i]);
	}
	
	printf("\nPARTITION FUNCTION TABLE:\n");
	printf("\n T9 = %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f\
 %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f",
		Tpf[0],Tpf[1],Tpf[2],Tpf[3],Tpf[4],Tpf[5],Tpf[6],Tpf[7],
		Tpf[8],Tpf[9],Tpf[10],Tpf[11],Tpf[12],Tpf[13],Tpf[14],Tpf[15],
		Tpf[16],Tpf[17],Tpf[18],Tpf[19],Tpf[20],Tpf[21],Tpf[22],Tpf[23]
	);
	for(int j=0; j<numberSpecies; j++){
		printf("\n%-5s %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f\
 %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f %4.2f",
	isoLabel[j],pf[j][0],pf[j][1],pf[j][2],pf[j][3],pf[j][4],pf[j][5],pf[j][6],pf[j][7],	
	pf[j][8],pf[j][9],pf[j][10],pf[j][11],pf[j][12],pf[j][13],pf[j][14],pf[j][15],
	pf[j][16],pf[j][17],pf[j][18],pf[j][19],pf[j][20],pf[j][21],pf[j][22],pf[j][23] );
	}
	
 	printf("\n");
}


/* Function to read rate parameter data file line by line, with filename as argument.
 This file is expected to have one reaction per line with the line structure
	 p0 p1 p2 p3 p4 p5 p6 reactionLabel
 where the pn are the values of the 7 Reaclib parameters for a reaction,
 reactionLabel is a label for the reaction that must contain no whitespace, and
 all fields on a line are separated by a blank space.
*/

void readLibraryParams (char *fileName)
{
	char line[120];
	char reaction[LABELSIZE];
	float p0, p1, p2, p3, p4, p5, p6, q, sf;
	int i0, i1, i2, i3, i4, i5, i6;
	int ii[6];

	// Open a file for reading  
	fr = fopen (fileName, "r");

	// Exit if the file doesn't exist or can't be read
	if( fr == NULL )
	{
		printf ("*** File Input Error: No readable file named %s\n", fileName);
		exit(1) ;
	}

	/* 
	Read in the file line by line and parse into variables.  The expected
	structure of each line is
	     float float float float float float float string
	each separated by a space, with no whitespace in the string.
	(See http://stackoverflow.com/questions/2854488/reading-a-string-with-spaces-with-sscanf
	for how to read string with spaces.)
	*/
	
	int n = -1;
	int subindex = -1;
	
	if(displayInput == 1) printf("\nData read in:\n\n");
	
	// Read in lines until NULL encountered. Lines can contain up to 120 characters
	
	while(fgets(line, 120, fr) != NULL)
	{
		subindex ++;
		
		switch(subindex){
			
			case 0:
				n++;
				sscanf (line, "%s %d %d %d %d %d %d %d %f %f", reaction, &i0, &i1, &i2, &i3, &i4, &i5, &i6, 
						&sf, &q);
				for(int j=0; j<LABELSIZE; j++){
					reacLabel[n][j] = reaction[j];
				}
				
				RGclass[n] = i0;
				RGmemberIndex[n] = i1;
				reaclibClass[n] = i2;
				NumReactingSpecies[n] = i3;
				NumProducts[n] = i4;
				isEC[n] = i5;
				isReverseR[n] = i6;
				Prefac[n] = sf;
				Q[n] = q;
				
				if(displayInput == 1) printf("\n\nReaction Index = %d",n);
				if(displayInput == 1) printf("\nisReverseR = %d reaclibIndex = %d",isReverseR[n],reaclibClass[n]);
				if(displayInput == 1) printf("\n%s %d %d %d %d %d %d %d %f %f", 
					reacLabel[n], 
		            RGclass[n],
					RGmemberIndex[n],
					reaclibClass[n],
					NumReactingSpecies[n],
		            NumProducts[n],
		            isEC[n],
					isReverseR[n],
					Prefac[n],
		            Q[n]
				);
				
			break;
			
			case 1:
				sscanf (line, "%f %f %f %f %f %f %f", &p0, &p1, &p2, &p3, &p4, &p5, &p6);
				P0[n] = p0;
				P1[n] = p1;
				P2[n] = p2;
				P3[n] = p3;
				P4[n] = p4;
				P5[n] = p5;
				P6[n] = p6;
				
				if(displayInput == 1) printf("\n%f %f %f %f %f %f %f", 
					P0[n], 
					P1[n],
					P2[n],
					P3[n],
		            P4[n],
		            P5[n],
					P6[n]
				);
				
			break;
			
			case 2:
				sscanf (line, "%d %d %d %d", &ii[0], &ii[1], &ii[2], &ii[3]);
				for(int mm=0; mm<NumReactingSpecies[n]; mm++)
				{
					reactantZ[n][mm] = ii[mm];
					if(displayInput == 1) printf("\n  Reactant[%d]: Z=%d", mm, reactantZ[n][mm]);
				}
				
			break;
			
			case 3:
				sscanf (line, "%d %d %d %d", &ii[0], &ii[1], &ii[2], &ii[3]);
				for(int mm=0; mm<NumReactingSpecies[n]; mm++)
				{
					reactantN[n][mm] = ii[mm];
					if(displayInput == 1) printf("\n  Reactant[%d]: N=%d", mm, reactantN[n][mm]);
				}
				
			break;
			
			case 4:
				sscanf (line, "%d %d %d %d", &ii[0], &ii[1], &ii[2], &ii[3]);
				for(int mm=0; mm<NumProducts[n]; mm++)
				{
					productZ[n][mm] = ii[mm];
					if(displayInput == 1) printf("\n  Product[%d]: Z=%d", mm, productZ[n][mm]);
				}
				
			break;
			
			case 5:
				sscanf (line, "%d %d %d %d", &ii[0], &ii[1], &ii[2], &ii[3]);
				for(int mm=0; mm<NumProducts[n]; mm++)
				{
					productN[n][mm] = ii[mm];
					if(displayInput == 1) printf("\n  Product[%d]: N=%d", mm, productN[n][mm]);
				}
				
			break;
			
			case 6:
				sscanf (line, "%d %d %d %d", &ii[0], &ii[1], &ii[2], &ii[3]);
				for(int mm=0; mm<NumReactingSpecies[n]; mm++)
				{
					ReactantIndex[n][mm] = ii[mm];
					if(displayInput == 1) printf("\n  ReactantIndex[%d]: N=%d", mm, ReactantIndex[n][mm]);
				}
				
			break;
			
			case 7:
				sscanf (line, "%d %d %d %d", &ii[0], &ii[1], &ii[2], &ii[3]);
				for(int mm=0; mm<NumProducts[n]; mm++)
				{
					ProductIndex[n][mm] = ii[mm];
					if(displayInput == 1) printf("\n  ProductIndex[%d]: N=%d", mm, ProductIndex[n][mm]);
				}
				subindex = -1;
				
			break;
					
		}
	
	}
	numberReactions = n+1;
	
	for(int i=0; i<numberReactions; i++)
	{
		Reactant1[i] = ReactantIndex[i][0];
		Reactant2[i] = ReactantIndex[i][1];
		Reactant3[i] = ReactantIndex[i][2];
	}
	
	fclose(fr);           // Close the file
}


/* Function to read the network data file line by line, with the filename as argument.
 This file is expected to have 4 lines per isotope with the line structure
	 isotopeSymbol A  Z  N  Y  MassExcess
	 pf00 pf01 pf02 pf03 pf04 pf05 pf06 pf07
	 pf10 pf11 pf12 pf13 pf14 pf15 pf16 pf17
	 pf20 pf21 pf22 pf23 pf24 pf25 pf26 pf27
where isotopeSymbol is an isotope label, A=Z+N is the atomic mass number, Z is the proton number, 
N is the neutron number, Y is the current abundance, MassExcess is the mass
excess in MeV, and the pf are 24 values of the partition function for that isotope at
different values of the temperature that will form a table for interpolation in temperature.
The assumed 24 values of the temperature for the partition function table are
{ 0.1, 0.15, 0.2, 0.3, 0.4, 0.5, 0.6, 0.7, 0.8, 0.9, 1.0, 1.5, 2.0, 2.5, 3.0, 3.5, 4.0, 
4.5, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0 } in units of 10^9 K.
All fields on a line are separated by a blank space and there is no whitespace in the isotopeSymbol.
The type signature of these four lines corresponding to a single isotope is
	string int int int float float
	float float float float float float float float
	float float float float float float float float
	float float float float float float float float
Here is an example for two isotopes:

ca40 40 20 20 0.0 -34.846
1.0 1.0 1.0 1.0 1.0 1.0 1.0 1.0
1.0 1.0 1.0 1.0 1.0 1.0 1.0 1.0
1.0 1.0 1.0 1.01 1.04 1.09 1.2 1.38
ti44 44 22 22 0.0 -37.548
1.0 1.0 1.0 1.0 1.0 1.0 1.0 1.0
1.0 1.0 1.0 1.0 1.01 1.03 1.08 1.14
1.23 1.35 1.49 1.85 2.35 3.01 3.86 4.94

A file with this format is written from the Java code to the file output/CUDAnetwork.inp using the
Java stream toCUDAnet.
	
*/

void readNetwork (char *fileName)
{
	char line[60];
	char isoSymbol[5];
	int z, n, a;
	float y, mass;
	float pf0, pf1, pf2, pf3, pf4, pf5, pf6, pf7;

	// Open a file for reading  
	fr = fopen (fileName, "r");

	// Exit if the file doesn't exist or can't be read
	if( fr == NULL )
	{
		printf ("*** File Input Error: No readable file named %s\n",fileName);
		exit(1) ;
	}
	
	// Read in the file line by line
	
	int isoIndex = -1;
	int isoSubIndex = 3;
	
	if(displayInput==1) printf("\nData read in:\n");
	
	// Read in lines until NULL encountered. Lines can contain up to 60 characters
	
	while(fgets(line, 60, fr) != NULL)
	{
		isoSubIndex ++;
		if(isoSubIndex == 4){
			isoSubIndex = 0;
			isoIndex ++;
			// Scan and parse a title line
			sscanf (line, "%s %d %d %d %f %f", isoSymbol, &a, &z, &n, &y, &mass);
			if(displayInput == 1)
			{
				printf("\n%s %d %d %d %f %f\n", isoSymbol, a, z, n, y, mass);
			}
			// Store variables in arrays
			Z[isoIndex] = z;
			N[isoIndex] = n;
			AA[isoIndex] = (float)a;
			Y[isoIndex] = y;
			X[isoIndex] = AA[isoIndex]*Y[isoIndex];
			massExcess[isoIndex] = mass;
			for(int j=0; j<5; j++){
				isoLabel[isoIndex][j] = isoSymbol[j];
			}
		} else {
			// Scan and parse a partition function line. 
			sscanf (line, "%f %f %f %f %f %f %f %f", &pf0, &pf1, &pf2, &pf3, &pf4, &pf5, &pf6, &pf7);
			if(displayInput == 1)
			{
				printf("%f %f %f %f %f %f %f %f\n", pf0, pf1, pf2, pf3, pf4, pf5, pf6, pf7);
			}
			// Store the partition function table values
			int tin = isoSubIndex-1;
			pf[isoIndex][8*(tin)] = pf0;
			pf[isoIndex][8*(tin)+1] = pf1;
			pf[isoIndex][8*(tin)+2] = pf2;
			pf[isoIndex][8*(tin)+3] = pf3;
			pf[isoIndex][8*(tin)+4] = pf4;
			pf[isoIndex][8*(tin)+5] = pf5;
			pf[isoIndex][8*(tin)+6] = pf6;
			pf[isoIndex][8*(tin)+7] = pf7;
		}
		
		numberSpecies = isoIndex + 1;
		
	}
}



// Function to test the CPU timer by executing a long, pointless loop.
void testTimerCPU()
{
	double a, b;
	
	START_CPU;
	for (long count = 1l; count < 500000l; count++) {
		a = sqrt(count);
		b = 1.0/logf(a);
		a = logf(b)/sqrt(a);
	}
	STOP_CPU;
	PRINT_CPU;	
}


/*
  Function to find the contributions to F+ and F- of each reaction for each isotope.  
  This is executed only once at the beginning of the entire calculation to determine 
  the structure of the network.
*/

void parseF()
{
	if(showParsing == 1)
		printf("\nUse parseF() to find F+ and F- flux components for each species:");
	
	int incrementPlus = 0;
	int incrementMinus = 0;
	
    // Loop over all isotopes in the network		
	for(int i=0; i<numberSpecies; i++)
	{
		int total = 0;
		int numFplus = 0;
		int numFminus = 0;
		if(showParsing == 1) printf("\n");
		
		// Loop over all possible reactions for this isotope, finding those that
		// change its population up (contributing to F+) or down (contributing
		// to F-).
		
		for(int j=0; j<numberReactions; j++)
		{
			int totalL = 0;
			int totalR = 0;
			
			// Loop over reactants for this reaction
			for(int k=0; k<NumReactingSpecies[j]; k++)
			{
				if(Z[i] == reactantZ[j][k] && N[i] == reactantN[j][k]) totalL ++;
			}
			
			// Loop over products for this reaction
			for(int k=0; k<NumProducts[j]; k++)
			{
				if(Z[i] == productZ[j][k] && N[i] == productN[j][k]) totalR ++;
			}
			
			total = totalL - totalR;
			
			if(total > 0)        // Contributes to F- for this isotope
			{
				numFminus ++;
				reacMask[i][j] = -total;
				tempInt2[incrementMinus + numFminus-1] = j;
				if(showParsing == 1)
				printf("\n%s reacIndex=%d %s nReac=%d nProd=%d totL=%d totR=%d tot=%d F-", 
					isoLabel[i], j, reacLabel[j], NumReactingSpecies[j], NumProducts[j], totalL, 
					totalR, total);
			} 
			else if(total < 0)   // Contributes to F+ for this isotope
			{
				numFplus ++;
				reacMask[i][j] = -total;
				tempInt1[incrementPlus + numFplus-1] = j;
				if(showParsing == 1)
				printf("\n%s reacIndex=%d %s nReac=%d nProd=%d totL=%d totR=%d tot=%d F+", 
					isoLabel[i], j, reacLabel[j], NumReactingSpecies[j], NumProducts[j], totalL, 
					totalR, total);
			}
			else                 // Does not contribute to flux for this isotope
			{
				reacMask[i][j] = 0;
			}
		}
		
		// Keep track of the total number of F+ and F- terms in the network for all isotopes
		totalFplus += numFplus;
		totalFminus += numFminus;
		
		numFluxPlus[i] = numFplus;
		numFluxMinus[i] = numFminus;
		
		incrementPlus += numFplus;
		incrementMinus += numFminus;
		
		if(showParsing == 1)
			printf("\n%d %s numF+ = %d numF- = %d", i, isoLabel[i], numFplus, numFminus);
	}
	
	// Display some cases
	
	printf("\n\nPART OF FLUX-ISOTOPE COMPONENT ARRAY (-n --> F-; +n --> F+ for given isotope):");
	
	if(numberSpecies != 16 && numberSpecies > 25)
	{
		// Comment out this part of the if-block for alpha network to prevent warnings about index being
		// out of bounds. (Doesn't matter in calculation since this block is not reached if it is an alpha
		// network with 16 species, but the compile generates a long string of warnings.)  Uncomment
		// to show up to 26 species for larger networks.
		
// 		printf("\n\nIndex               Reaction%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s\
// %5s%5s%5s%5s%5s%5s",
// 			isoLabel[0], isoLabel[1], isoLabel[2], isoLabel[3], isoLabel[4], isoLabel[5], isoLabel[6],
// 			isoLabel[7], isoLabel[8], isoLabel[9], isoLabel[10], isoLabel[11], isoLabel[12], isoLabel[13],
// 			isoLabel[14], isoLabel[15], isoLabel[16], isoLabel[17], isoLabel[18], isoLabel[19],
// 			isoLabel[20], isoLabel[21], isoLabel[22], isoLabel[23], isoLabel[24], isoLabel[25]
// 		);
// 		for(int j=0; j<numberReactions; j++)
// 		{
// 			
// 			printf(
// 				"\n %4d %22s %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d\
//  %4d %4d %4d %4d %4d %4d",
// 				j, reacLabel[j], reacMask[0][j], reacMask[1][j], reacMask[2][j],
// 				reacMask[3][j], reacMask[4][j], reacMask[5][j], reacMask[6][j], 
// 				reacMask[7][j], reacMask[8][j], reacMask[9][j], reacMask[10][j],
// 				reacMask[11][j], reacMask[12][j], reacMask[13][j], reacMask[14][j],
// 				reacMask[15][j], reacMask[16][j], reacMask[17][j], reacMask[18][j],
// 				reacMask[19][j], reacMask[20][j], reacMask[21][j], reacMask[22][j],
// 				reacMask[23][j], reacMask[24][j], reacMask[25][j]
// 			);
// 		}

	} 
	else if(numberSpecies > 15)  // For alpha networks
	{
		printf("\n\nIndex               Reaction%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s",
			isoLabel[0], isoLabel[1], isoLabel[2], isoLabel[3], isoLabel[4], isoLabel[5], isoLabel[6],
			isoLabel[7], isoLabel[8], isoLabel[9], isoLabel[10], isoLabel[11], isoLabel[12], isoLabel[13],
			isoLabel[14], isoLabel[15]	
		);
		for(int j=0; j<numberReactions; j++)
		{
			
			printf(
				"\n %4d %22s %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d %4d",
				j, reacLabel[j], reacMask[0][j], reacMask[1][j], reacMask[2][j],
				reacMask[3][j], reacMask[4][j], reacMask[5][j], reacMask[6][j], 
				reacMask[7][j], reacMask[8][j], reacMask[9][j], reacMask[10][j],
				reacMask[11][j], reacMask[12][j], reacMask[13][j], reacMask[14][j],
				reacMask[15][j]
			);
		}
	}
	
	printf("\n\nFLUX SPARSENESS: Non-zero F+ = %d; Non-zero F- = %d, out of %d x %d = %d possibilities.", 
		totalFplus, totalFminus, SIZE, ISOTOPES, SIZE*ISOTOPES);
}


// Function to check that a valid device exists. Copied from
// http://www.ncsa.illinois.edu/UserInfo/Training/Workshops/CUDA/presentations/tutorial-CUDA.html

void devcheck(int gpudevice) 
{ 
	int device_count=0; 
	int device;  // used with  cudaGetDevice() to verify cudaSetDevice() 

	// get the number of non-emulation devices  detected 
	hipGetDeviceCount( &device_count); 
	if (gpudevice > device_count) 
	{ 
		printf("gpudevice >=  device_count ... exiting\n"); 
		exit(1); 
	} 
	hipError_t cudareturn; 
	hipDeviceProp_t deviceProp; 
    
	// cudaGetDeviceProperties() is also  demonstrated in the deviceQuery/ example
	// of the sdk projects directory 
	
	hipGetDeviceProperties(&deviceProp,  gpudevice); 
	printf("\n[deviceProp.major.deviceProp.minor] = [%d.%d]\n", 
	deviceProp.major, deviceProp.minor); 

	if (deviceProp.major > 999) 
	{ 
		printf("warning, CUDA Device  Emulation (CPU) detected, exiting\n"); 
		exit(1); 
	} 
   
	// choose a cuda device for kernel  execution 
	cudareturn=hipSetDevice(gpudevice); 
	if (cudareturn == hipErrorInvalidDevice) 
	{ 
		perror("hipSetDevice returned  hipErrorInvalidDevice"); 
	} 
	else 
	{ 
		// double check that device was properly selected 
		hipGetDevice(&device); 
		printf("cudaGetDevice()=%d\n",device); 
	} 
}
